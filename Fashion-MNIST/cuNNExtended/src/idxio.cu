#define _AMD64_ // architecture
#define WIN32_LEAN_AND_MEAN
#define WIN32_EXTRA_MEAN
#include <cstdbool>
#include <cstdint>
#include <cstdio>
#include <vector>

#include <errhandlingapi.h>
#include <fileapi.h>
#include <handleapi.h>
#include <heapapi.h>
#include <windef.h>

// a generic file reading routine, that reads in an existing binary file and returns the buffer. (NULL in case of a failure)
// returned memory needs to be freed using HeapFree()! NOT UCRT's free()
static inline uint8_t* open(_In_ const wchar_t* const file_name, _Inout_ size_t* const size) {
    uint8_t*       buffer    = nullptr;
    DWORD          nbytes    = 0UL;
    LARGE_INTEGER  liFsize   = { .QuadPart = 0LLU };
    const HANDLE64 hFile     = ::CreateFileW(file_name, GENERIC_READ, 0, nullptr, OPEN_EXISTING, FILE_ATTRIBUTE_READONLY, nullptr);

    // process's default heap, non serializeable
    const HANDLE64 hProcHeap = ::GetProcessHeap();

    if (hFile == INVALID_HANDLE_VALUE) {
        fwprintf_s(stderr, L"Error %lu in CreateFileW\n", ::GetLastError());
        goto INVALID_HANDLE_ERR;
    }

    if (!hProcHeap) {
        fwprintf_s(stderr, L"Error %lu in GetProcessHeap\n", ::GetLastError());
        goto GET_FILESIZE_ERR;
    }

    if (!::GetFileSizeEx(hFile, &liFsize)) {
        fwprintf_s(stderr, L"Error %lu in GetFileSizeEx\n", ::GetLastError());
        goto GET_FILESIZE_ERR;
    }

    if (!(buffer = static_cast<uint8_t*>(::HeapAlloc(hProcHeap, 0UL, liFsize.QuadPart)))) {
        fwprintf_s(stderr, L"Error %lu in HeapAlloc\n", ::GetLastError());
        goto GET_FILESIZE_ERR;
    }

    if (!ReadFile(hFile, buffer, liFsize.QuadPart, &nbytes, nullptr)) {
        fwprintf_s(stderr, L"Error %lu in ReadFile\n", ::GetLastError());
        goto READFILE_ERR;
    }

    ::CloseHandle(hFile);
    *size = liFsize.QuadPart;
    return buffer;

READFILE_ERR:
    ::HeapFree(hProcHeap, 0UL, buffer);
GET_FILESIZE_ERR:
    ::CloseHandle(hFile);
INVALID_HANDLE_ERR:
    *size = 0;
    return nullptr;
}

// a file format agnostic write routine to serialize binary image files.
// if a file with the specified name exists on disk, it will be overwritten.
static inline bool serialize(
    _In_ const wchar_t* const filename,
    _In_ const uint8_t* const buffer,
    _In_ const size_t         size,
    _In_ const bool           freebuffer /* specifies whether to free the buffer after serialization */
) {
    // buffer is assumed to be allocated with HeapAlloc, i.e HeapFree will be invoked to free the buffer NOT UCRT's free()
    // one major caveat is that the caller needs to pass in a byte stream instead of a image struct, which implies a potentially
    // unnecessary memory allocationand buffer creation from the image structs.
    // defining separate write routines for each image format will be redundant and will ruin the modularity of the project.

    const HANDLE64 hFile  = ::CreateFileW(filename, GENERIC_WRITE, 0, nullptr, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL, nullptr);
    DWORD          nbytes = 0;

    if (hFile == INVALID_HANDLE_VALUE) {
        fwprintf_s(stderr, L"Error %4lu in CreateFileW\n", ::GetLastError());
        goto PREMATURE_RETURN;
    }

    if (!WriteFile(hFile, buffer, size, &nbytes, nullptr)) {
        fwprintf_s(stderr, L"Error %4lu in WriteFile\n", ::GetLastError());
        goto PREMATURE_RETURN;
    }

    ::CloseHandle(hFile);
    if (freebuffer) {
        const HANDLE64 hProcHeap = GetProcessHeap(); // WARNING :: ignoring potential errors here
        ::HeapFree(hProcHeap, 0, buffer);
    }
    ::CloseHandle(hFile);
    return true;

PREMATURE_RETURN:
    ::CloseHandle(hFile);
    return false;
}
